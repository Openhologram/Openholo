#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install, copy or use the software.
//
//
//                           License Agreement
//                For Open Source Digital Holographic Library
//
// Openholo library is free software;
// you can redistribute it and/or modify it under the terms of the BSD 2-Clause license.
//
// Copyright (C) 2017-2024, Korea Electronics Technology Institute. All rights reserved.
// E-mail : contact.openholo@gmail.com
// Web : http://www.openholo.org
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//  1. Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//  2. Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the copyright holder or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
// This software contains opensource software released under GNU Generic Public License,
// NVDIA Software License Agreement, or CUDA supplement to Software License Agreement.
// Check whether software you use contains licensed software.
//
//M*/

/**
* @file		ophPCKernel.cu
* @brief	Openholo Point Cloud based CGH generation with CUDA GPGPU
* @author	Hyeong-Hak Ahn
* @date		2018/09
*/

#ifndef OphPCKernel_cu__
#define OphPCKernel_cu__

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>

#include "typedef.h"
#include "ophPointCloud_GPU.h"


__global__ void cudaKernel_diffractEncodedRS(Real* pc_data, Real* amp_data, const GpuConstERS* config, const int n_points_stream, Real* dst) {
	ulonglong tid = blockIdx.x * blockDim.x + threadIdx.x;
	ulonglong tid_offset = blockDim.x * gridDim.x;
	ulonglong n_pixels = config->pn_X * config->pn_Y;

	for (tid; tid < n_pixels; tid += tid_offset) {
		int xxtr = tid % config->pn_X;
		int yytr = tid / config->pn_X;
		ulonglong idx = xxtr + yytr * config->pn_X;

		Real xxx = ((Real)xxtr + 0.5) * config->pp_X - config->half_ss_X;
		Real yyy = config->half_ss_Y - ((Real)yytr + 0.5) * config->pp_Y;
		Real interWav = xxx * config->sin_thetaX + yyy * config->sin_thetaY;

		for (int j = 0; j < n_points_stream; ++j) { //Create Fringe Pattern
			Real pcx = pc_data[3 * j + _X] * config->scale_X;
			Real pcy = pc_data[3 * j + _Y] * config->scale_Y;
			Real pcz = pc_data[3 * j + _Z] * config->scale_Z + config->offset_depth;

			Real r = sqrt((xxx - pcx) * (xxx - pcx) + (yyy - pcy) * (yyy - pcy) + (pcz * pcz));
			Real p = config->k * (r - interWav);
			Real res = amp_data[config->n_colors * j] * cos(p);

			*(dst + idx) += res;
		}
	}
	__syncthreads();
}


__global__ void cudaKernel_diffractNotEncodedRS(Real* pc_data, Real* amp_data, const GpuConstNERS* config, const int n_points_stream, Real* dst_real, Real* dst_imag) {
	ulonglong tid = blockIdx.x * blockDim.x + threadIdx.x;
	ulonglong tid_offset = blockDim.x * gridDim.x;
	ulonglong n_pixels = config->pn_X * config->pn_Y;

	for (tid; tid < n_pixels; tid += tid_offset) {
		int xxtr = tid % config->pn_X;
		int yytr = tid / config->pn_X;
		ulonglong idx = xxtr + yytr * config->pn_X;

		Real xxx = -config->half_ss_X + (xxtr - 1) * config->pp_X;
		Real yyy = -config->half_ss_Y + (config->pn_Y - yytr) * config->pp_Y;

		for (int j = 0; j < n_points_stream; ++j) { //Create Fringe Pattern
			Real pcx = pc_data[3 * j + _X] * config->scale_X;
			Real pcy = pc_data[3 * j + _Y] * config->scale_Y;
			Real pcz = pc_data[3 * j + _Z] * config->scale_Z + config->offset_depth;
			Real amplitude = amp_data[config->n_colors * j];

			//boundary test
			Real abs_det_txy_pcz = abs(config->det_tx * pcz);
			Real _xbound[2] = {
				pcx + abs_det_txy_pcz,
				pcx - abs_det_txy_pcz
			};

			abs_det_txy_pcz = abs(config->det_ty * pcz);
			Real _ybound[2] = {
				pcy + abs_det_txy_pcz,
				pcy - abs_det_txy_pcz
			};

			Real Xbound[2] = {
				floor((_xbound[0] + config->half_ss_X) / config->pp_X) + 1,
				floor((_xbound[1] + config->half_ss_X) / config->pp_X) + 1
			};

			Real Ybound[2] = {
				config->pn_Y - floor((_ybound[1] + config->half_ss_Y) / config->pp_Y),
				config->pn_Y - floor((_ybound[0] + config->half_ss_Y) / config->pp_Y)
			};

			if (Xbound[0] > config->pn_X)	Xbound[0] = config->pn_X;
			if (Xbound[1] < 0)				Xbound[1] = 0;
			if (Ybound[0] > config->pn_Y)	Ybound[0] = config->pn_Y;
			if (Ybound[1] < 0)				Ybound[1] = 0;
			//

			if (((xxtr >= Xbound[1]) && (xxtr < Xbound[0])) && ((yytr >= Ybound[1]) && (yytr < Ybound[0]))) {
				Real xxx_pcx_sq = (xxx - pcx) * (xxx - pcx);
				Real yyy_pcy_sq = (yyy - pcy) * (yyy - pcy);
				Real pcz_sq = pcz * pcz;

				//range test
				Real abs_det_txy_sqrt = abs(config->det_tx * sqrt(yyy_pcy_sq + pcz_sq));
				Real range_x[2] = {
					pcx + abs_det_txy_sqrt,
					pcx - abs_det_txy_sqrt
				};

				abs_det_txy_sqrt = abs(config->det_ty * sqrt(xxx_pcx_sq + pcz_sq));
				Real range_y[2] = {
					pcy + abs_det_txy_sqrt,
					pcy - abs_det_txy_sqrt
				};
				//

				if (((xxx < range_x[0]) && (xxx > range_x[1])) && ((yyy < range_y[0]) && (yyy > range_y[1]))) {
					Real r = sqrt(xxx_pcx_sq + yyy_pcy_sq + pcz_sq);
					Real p = config->k * r;
					Real a = (amplitude * pcz) / (config->lambda * r * r);;
					Real res_real = sin(p) * a;
					Real res_imag = -cos(p) * a;

					*(dst_real + idx) += res_real;
					*(dst_imag + idx) += res_imag;
				}
			}
		}
	}
	__syncthreads();
}


__global__ void cudaKernel_diffractNotEncodedFrsn(Real* pc_data, Real* amp_data, const GpuConstNEFR* config, const int n_points_stream, Real* dst_real, Real* dst_imag) {
	ulonglong tid = blockIdx.x * blockDim.x + threadIdx.x;
	ulonglong tid_offset = blockDim.x * gridDim.x;
	ulonglong n_pixels = config->pn_X * config->pn_Y;

	for (tid; tid < n_pixels; tid += tid_offset) {
		int xxtr = tid % config->pn_X;
		int yytr = tid / config->pn_X;
		ulonglong idx = xxtr + yytr * config->pn_X;

		Real xxx = -config->half_ss_X + (xxtr - 1) * config->pp_X;
		Real yyy = -config->half_ss_Y + (config->pn_Y - yytr) * config->pp_Y;

		for (int j = 0; j < n_points_stream; ++j) { //Create Fringe Pattern
			Real pcx = pc_data[3 * j + _X] * config->scale_X;
			Real pcy = pc_data[3 * j + _Y] * config->scale_Y;
			Real pcz = pc_data[3 * j + _Z] * config->scale_Z + config->offset_depth;
			Real amplitude = amp_data[config->n_colors * j];

			//boundary test
			Real abs_txy_pcz = abs(config->tx * pcz);
			Real _xbound[2] = {
				pcx + abs_txy_pcz,
				pcx - abs_txy_pcz
			};

			abs_txy_pcz = abs(config->ty * pcz);
			Real _ybound[2] = {
				pcy + abs_txy_pcz,
				pcy - abs_txy_pcz
			};

			Real Xbound[2] = {
				floor((_xbound[0] + config->half_ss_X) / config->pp_X) + 1,
				floor((_xbound[1] + config->half_ss_X) / config->pp_X) + 1
			};

			Real Ybound[2] = {
				config->pn_Y - floor((_ybound[1] + config->half_ss_Y) / config->pp_Y),
				config->pn_Y - floor((_ybound[0] + config->half_ss_Y) / config->pp_Y)
			};

			if (Xbound[0] > config->pn_X)	Xbound[0] = config->pn_X;
			if (Xbound[1] < 0)				Xbound[1] = 0;
			if (Ybound[0] > config->pn_Y)	Ybound[0] = config->pn_Y;
			if (Ybound[1] < 0)				Ybound[1] = 0;
			//

			if (((xxtr >= Xbound[1]) && (xxtr < Xbound[0])) && ((yytr >= Ybound[1]) && (yytr < Ybound[0]))) {
				Real p = config->k * ((xxx - pcx) * (xxx - pcx) + (yyy - pcy) * (yyy - pcy) + (2 * pcz * pcz)) / (2 * pcz);
				Real a = amplitude / (config->lambda * pcz);
				Real res_real = sin(p) * a;
				Real res_imag = -cos(p) * a;

				*(dst_real + idx) += res_real;
				*(dst_imag + idx) += res_imag;
			}
		}
	}
	__syncthreads();
}


extern "C"
{
	void cudaGenCghPointCloud_EncodedRS(
		const int &nBlocks, const int &nThreads, const int &n_pts_per_stream,
		Real* cuda_pc_data, Real* cuda_amp_data,
		Real* cuda_dst,
		const GpuConstERS* cuda_config)
	{
		cudaKernel_diffractEncodedRS << < nBlocks, nThreads >> > (cuda_pc_data, cuda_amp_data, cuda_config, n_pts_per_stream, cuda_dst);
	}

	void cudaGenCghPointCloud_NotEncodedRS(
		const int &nBlocks, const int &nThreads, const int &n_pts_per_stream,
		Real* cuda_pc_data, Real* cuda_amp_data,
		Real* cuda_dst_real, Real* cuda_dst_imag,
		const GpuConstNERS* cuda_config)
	{
		cudaKernel_diffractNotEncodedRS << < nBlocks, nThreads >> > (cuda_pc_data, cuda_amp_data, cuda_config, n_pts_per_stream, cuda_dst_real, cuda_dst_imag);
	}

	void cudaGenCghPointCloud_NotEncodedFrsn(
		const int &nBlocks, const int &nThreads, const int &n_pts_per_stream,
		Real* cuda_pc_data, Real* cuda_amp_data,
		Real* cuda_dst_real, Real* cuda_dst_imag,
		const GpuConstNEFR* cuda_config)
	{
		cudaKernel_diffractNotEncodedFrsn << < nBlocks, nThreads >> > (cuda_pc_data, cuda_amp_data, cuda_config, n_pts_per_stream, cuda_dst_real, cuda_dst_imag);
	}
}

#endif // !OphPCKernel_cu__